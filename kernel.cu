#include "hip/hip_runtime.h"
// Pour lancer un programme et utiliser le profiler, lancer le cmd en admin
// astuce: win + R, cmd, Ctr + Shift + enter
// Ici, f8 build, f9 run, f10 profile


#include "kernel.h"
#include <iostream>
#include <math.h>
#include <fstream>


__global__
void mandelbrot(int N, int* z, int W, int H, double xmin, double xmax, double ymin, double ymax, int iter) {

	double c_re, c_im;
	double z_re = 0.0;
	double z_im = 0.0;
	double z_re2, z_im2;

	int i, j;

	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	const int stride = blockDim.x * gridDim.x;

	int k;
	for (k = index; k < N; k += stride) {
		i = k / W;
		j = k % W;
		c_re = xmin + (double)(xmax - xmin) * (double)j / W;
		c_im = ymin + (double)(ymax - ymin) * (double)i / H;

		bool verif = true;
		for (int it = 0; it < iter; ++it) {
			z_re2 = z_re * z_re;
			z_im2 = z_im * z_im;

			z_im = 2 * z_re * z_im + c_im;
			z_re = z_re2 - z_im2 + c_re;

			if (z_re2 + z_im2 > 4.0) {
				z[k] = it *255 / iter;
				verif = false;
				break;
			}
		}
		if (verif == true) {
			z[k] = 0;
		}
	}
}



void newFrame(int* z, int W, int H, double xmin, double xmax, double ymin, double ymax , int iter =255) {
	const unsigned int N = H * W;

	// alloc
	int* d_z; // limit � faire en passage par r�f�rence
	hipMalloc(&d_z, N * sizeof(int));
	hipMemcpy(d_z, z, N * sizeof(double), hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	mandelbrot <<<numBlocks, blockSize >>> (N, d_z, W, H, xmin, xmax, ymin, ymax, iter);
	hipMemcpy(z, d_z, N * sizeof(int), hipMemcpyDeviceToHost);

	//free (attention le z n'a pas �t� free)
	hipFree(d_z);
}