#include "hip/hip_runtime.h"
// Pour lancer un programme et utiliser le profiler, lancer le cmd en admin
// astuce: win + R, cmd, Ctr + Shift + enter
// Ici, f8 build, f9 run, f10 profile


#include "saver.h"
#include <iostream>
#include <math.h>
#include <fstream>


__global__
void mandelbrot(int N, int* z, int W, int H, double xmin, double xmax, double ymin, double ymax, int iter = 100) {

	double c_re, c_im;
	double z_re = 0.0;
	double z_im = 0.0;
	double z_re2, z_im2;

	int i, j;

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	int k;
	for (k = index; k < N; k += stride) {
		i = k / W;
		j = k % W;
		c_re = xmin + (double)(xmax - xmin) * (double)j / W;
		c_im = ymin + (double)(ymax - ymin) * (double)i / H;

		bool verif = true;
		for (int it = 0; it < iter; ++it) {
			z_re2 = z_re * z_re;
			z_im2 = z_im * z_im;

			z_im = 2 * z_re * z_im + c_im;
			z_re = z_re2 - z_im2 + c_re;

			if (z_re2 + z_im2 > 4.0) {
				z[k] = it;
				verif = false;
				break;
			}
		}
		if (verif == true) {
			z[k] = 0;
		}
	}
}


int main(void)
{

	//number of items on each axiss
	int W = 5'000;
	int H = 5'000;
	int N = H * W;

	double xmin = -2.0;
	double xmax = 0.5;
	double ymin = -1.25;
	double ymax = 1.25;

	int iter = 100;

	std::cout << W << " " << H << std::endl;
	int* z, * d_z;

	z = (int*)malloc(N * sizeof(int));
	hipMalloc(&d_z, N * sizeof(int));

	hipMemcpy(d_z, z, N * sizeof(double), hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;

	std::cout << "avant le lancement du kernel" << std::endl;
	mandelbrot << <numBlocks, blockSize >> > (N, d_z, W, H, xmin, xmax, ymin, ymax, iter);
	std::cout << "apr�s le lancement du kernel" << std::endl;
	std::cout << "avant la copie" << std::endl;
	hipMemcpy(z, d_z, N * sizeof(int), hipMemcpyDeviceToHost);
	std::cout << "apr�s la copie" << std::endl;




	save(z, W, H);

	/*
	std::ofstream file;
	std::cout << "D�but de l'�criture" << std::endl;
	file.open("fractal_parallel.pnm");

	std::cout << "After open" << std::endl;
	file << "P3" << "\n";
	file << W << " " << H << "\n";
	file << iter << std::endl;
	for (int i = 0; i < H; i++) {
		for (int j = 0; j < W; j++) {
			file << "  " << z[i * H + j] << "  " << z[i * H + j] << "  " << z[i * H + j] << "  \n";
		}
	}

	file.close();

	std::cout << "Fin de l'�criture" << "\n";
	hipFree(d_z);
	free(z);
	std::cout << "Fin des free" << "\n";
	*/
	return 0;
}
